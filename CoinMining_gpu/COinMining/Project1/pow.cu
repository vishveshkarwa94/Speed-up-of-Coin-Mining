#include "hip/hip_runtime.h"
#include <iostream>
#include <string> 
#include <iomanip>
#include <math.h>
#include <sstream>
#include "sha256.h"
#include "sha256.cuh"

using namespace std;

__managed__ int nonce;

__global__
void pow(string initBlockHash, string initTargetHash, int range) {

	cout<<"Performing Proof-of-Work...wait..."<<endl;
	// index = block index * number of threads per block + thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // stride  = number threads per block * number of block per grid
    int stride = blockDim.x * gridDim.x;
	nonce = 0;
	string tmp_hash = "undefined";
	for (int tmp_nonce = index; tmp_nonce <= range; tmp_nonce= tmp_nonce+stride) {
		//cout << " Checking for Nonce : " << nonce << endl;
		tmp_hash = sha256(sha256(initBlockHash+ std::to_string(tmp_nonce)));
		if (initTargetHash.compare(tmp_hash) > 0) {
			nonce = tmp_nonce;
			break;
		}
	}
}
int main() {

	string initBlockHash = sha256("CSCI-654 Foundations of Parallel Computing");
	cout << initBlockHash << endl;
	string initTargetHash = "0000092a6893b712892a41e8438e3ff2242a68747105de0395826f60b38d88dc";
	int nonce = 0;	
	int range = INT_MAX - INT_MIN;
	int blockSize = 256;
	int numBlocks = (range + blockSize - 1) / blockSize;
	pow <<<numBlocks, blockSize >>> (initBlockHash,initTargetHash,range);
	hipDeviceSynchronize();
	cout << "Nonce : " << nonce << endl;
}